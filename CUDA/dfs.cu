#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <queue>
#include <sstream>
#include <hip/hip_runtime.h>
#define NODES 67108863            //Only 2 exp(n)-1 values

//This Works Only On Cpmplete Binary Tree
//You Can Use a Non-Complete Tree if you complete nodes with NULL

using namespace std;

typedef struct node{
    bool visited;
    int n;
    struct node *left;
    struct node *right;
    struct node *parent;
}node;

__constant__ node *tree;

void toArray(node **tree, int nodes){
    *tree = (node*)malloc(nodes * sizeof(**tree));
}

//On Huge Trees:: can be paralelized
__global__ void makeEdges(int length){
    int limit = (length/2) - 1;
    int x = threadIdx.x;
    //printf("Value on limit index: %d \n", tree[limit].n);
    //for(int x = 0; x <= limit; x++){
    if(x < limit){
        tree[x].left = &tree[(2 * x) + 1];
        tree[x].right = &tree[(2 * x) + 2];
    }
    /*for(int y = (limit + 1); y < NODES; y++){
        tree[y].left = NULL;
        tree[y].right = NULL;
    }*/
}

//On Huge Trees:: can be paralelized
__global__ void findParents(int nodes){
    int x = threadIdx.x;
    if(x < nodes){
        if(x == 0){
            tree[x].parent = NULL;
        }else{
            int par = (x - 1)/2;
            tree[x].parent = &tree[par];
        }
    }
}

//On Huge Trees:: can be paralelized
__global__ void makeTree(int nodes){
    int x = threadIdx.x;
    if(x < nodes){
        tree[x].n = x + 1;
        tree[x].visited = false;
    }
}

bool hasChildren(node n){
    if(n.left != NULL && n.right != NULL){
        return true;
    }else{
        return false;
    }
}

//The Interesting Part!!
void DFS(int element, node* tree){
    bool found = false;
    queue <int> path;
    node *temp = &tree[0];

	do{
	    //printf("On node with element %d \n", temp->n);
	    if(temp->n != element){
	    	path.push(temp->n);
	        if (hasChildren(*temp)){
	            if(temp->left->visited == false){
	                temp = temp->left;
	            }else if(temp->right->visited == false){
					temp = temp->right;
				}else{
					temp->visited = true;
					temp = temp->parent;
				}
	        }else{
	            temp->visited = true;
	            temp = temp->parent;
	        }
	    }else{
	        found = true;
	        path.push(temp->n);
	        printf("Element Found!: %d \n", temp->n);
	    }
	}while(found == false);
/*    while(!path.empty()){
		printf(" %d ", path.front());
		path.pop();
	}
*/
	printf("\n");
}

int main(int argc, char **argv){
	int to_find = 0;
	stringstream ss;
    //Creation of Tree Array
    toArray(&tree, NODES);
    hipMalloc((node**)&tree, (NODES*sizeof(node)));
    //makeTree(NODES);
    makeTree<<<1,1>>>(NODES);
    //hipMemcpy(&tree, d_tree, hipMemcpyDeviceToHost);
    //END
    /*for(int i; i < NODES; i++){
        printf("Node %d element: %d \n", i, tree[i].n);
    }*/
    //printf("Levels %d \n", lvlNumber(NODES));
    //printf("Last Level Nodes %d \n", lastLvlNodes(lvlNumber(NODES)));
    //makeEdges(NODES);
    makeEdges<<<1,1>>>(NODES);
    //hipMemcpy(d_tree, &tree, hipMemcpyHostToDevice);
    //findParents();
    findParents<<<1,1>>>(NODES);
    //printf("Right value from 2 node's child: %d \n", tree[2].right->n);
    //printf("Left value from 2 node's child: %d \n", tree[2].left->n);
    //printf("Parent of node 14 element: %d \n", tree[14].parent->n);
    //printf("Has the node 14 children?: %d \n", hasChildren(tree[14]));
    hipMemcpy(&ar, &tree, (NODES*sizeof(*tree)),hipMemcpyDeviceToHost);

    ss << argv[1];
    ss >> to_find;
    DFS(to_find);
    hipFree(tree);
    return(0);
}
